#include "hip/hip_runtime.h"
#include "gpu_volume.h"

#include "stk/cuda/hip/hip_runtime.h"
#include "stk/cuda/ptr.h"

#include <algorithm>

namespace cuda = stk::cuda;

__global__ void reduce_volume_min_max(
    const cuda::VolumePtr<float> in,
    dim3 dims,
    float2* out)
{
    extern __shared__ float2 shared[];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    shared[tid].x = FLT_MAX;
    shared[tid].y = -FLT_MAX;

    if (x < dims.x &&
        y < dims.y &&
        z < dims.z) {
        shared[tid].x = in(x,y,z);
        shared[tid].y = in(x,y,z);
    }
    __syncthreads();

    if (tid < 256) { 
        shared[tid].x = min(shared[tid].x, shared[tid+256].x);
        shared[tid].y = max(shared[tid].y, shared[tid+256].y);
    }
    __syncthreads();

    if (tid < 128) {
        shared[tid].x = min(shared[tid].x, shared[tid+128].x);
        shared[tid].y = max(shared[tid].y, shared[tid+128].y);
    }
    __syncthreads();

    if (tid < 64) {
        shared[tid].x = min(shared[tid].x, shared[tid+64].x);
        shared[tid].y = max(shared[tid].y, shared[tid+64].y);
    }
    __syncthreads();

    if (tid < 32) {
        shared[tid].x = min(shared[tid].x, shared[tid+32].x);
        shared[tid].y = max(shared[tid].y, shared[tid+32].y);
    }
    __syncthreads();

    if (tid < 16) {
        shared[tid].x = min(shared[tid].x, shared[tid+16].x);
        shared[tid].y = max(shared[tid].y, shared[tid+16].y);
    }
    __syncthreads();
    
    if (tid < 8) {
        shared[tid].x = min(shared[tid].x, shared[tid+8].x);
        shared[tid].y = max(shared[tid].y, shared[tid+8].y);
    }
    __syncthreads();
    
    if (tid < 4) {
        shared[tid].x = min(shared[tid].x, shared[tid+4].x);
        shared[tid].y = max(shared[tid].y, shared[tid+4].y);
    }
    __syncthreads();

    if (tid < 2) {
        shared[tid].x = min(shared[tid].x, shared[tid+2].x);
        shared[tid].y = max(shared[tid].y, shared[tid+2].y);
    }
    __syncthreads();
    
    if (tid == 0) {
        // Write min/max for block to output
        out[bid].x = min(shared[tid].x, shared[tid+1].x);
        out[bid].y = max(shared[tid].y, shared[tid+1].y);
    }
}

__global__ void reduce_min_max(
    unsigned int n,
    float2* in,
    float2* out)
{
    extern __shared__ float2 shared[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;

    if (i < n) shared[tid] = in[i];
    else shared[tid] = {FLT_MAX, -FLT_MAX};

    __syncthreads();

    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        if ((tid % (2*s)) == 0)
        {
            shared[tid].x = min(shared[tid].x, shared[tid + s].x);
            shared[tid].y = max(shared[tid].y, shared[tid + s].y);
        }

        __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = shared[0];
}

namespace stk {
    void find_min_max(const GpuVolume& vol, float& min, float& max)
    {
        dim3 block_size{8,8,8};

        dim3 grid_size {
            (vol.size().x + block_size.x - 1) / block_size.x,
            (vol.size().y + block_size.y - 1) / block_size.y,
            (vol.size().z + block_size.z - 1) / block_size.z
        };

        // Number of blocks (or values in the active buffer)
        uint32_t n = grid_size.x * grid_size.y * grid_size.z;

        // Allocate our global buffers
        float2* d_out;
        CUDA_CHECK_ERRORS(hipMalloc(&d_out, 2*n*sizeof(float)));

        float2* d_in;
        CUDA_CHECK_ERRORS(hipMalloc(&d_in, 2*n*sizeof(float)));

        reduce_volume_min_max<<<grid_size, block_size, 
            uint32_t(2*sizeof(float)*512)>>>(
            vol, vol.size(), d_out
        );

        CUDA_CHECK_ERRORS(hipPeekAtLastError());
        CUDA_CHECK_ERRORS(hipDeviceSynchronize());

        while (n > 1) {
            // block_count should always be pow2 as it follows the gridsize from 
            //  previous step
            uint32_t n_threads = std::min<uint32_t>(n, 1024);
            uint32_t n_blocks = (n + n_threads - 1) / n_threads;

            CUDA_CHECK_ERRORS(hipMemcpy(d_in, d_out, 2*n*sizeof(float), 
                hipMemcpyDeviceToDevice));

            reduce_min_max<<<{n_blocks,1,1}, {n_threads,1,1}, 
                            uint32_t(2*sizeof(float)*n_threads)>>>(
                n, d_in, d_out);

            CUDA_CHECK_ERRORS(hipPeekAtLastError());
            CUDA_CHECK_ERRORS(hipDeviceSynchronize());

            n = n_blocks;
        }

        float2 min_max;
        CUDA_CHECK_ERRORS(hipMemcpy(&min_max, d_out, 2*sizeof(float), hipMemcpyDeviceToHost));

        min = min_max.x;
        max = min_max.y;

        CUDA_CHECK_ERRORS(hipFree(d_in));
        CUDA_CHECK_ERRORS(hipFree(d_out));
    }
}