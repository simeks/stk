#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <hip/hip_runtime.h>

#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/volume.h>
#include <stk/image/gpu_volume.h>
#include <stk/image/volume.h>

#include "test_util.h"

using namespace stk;

template<typename T>
__global__ void linear_at_border_kernel(cuda::VolumePtr<T> in, cuda::VolumePtr<T> out, dim3 dims, float3 offset)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= dims.x || y >= dims.y || z >= dims.z) {
        return;
    }

    out(x,y,z) = cuda::linear_at_border(in, dims, x+offset.x, y+offset.y, z+offset.z);
}

TEST_CASE("cuda_linear_at", "[cuda]")
{
    dim3 dims{2,2,2};
    
    VolumeFloat in(dims, 1.0f);
    in(0,0,0) = 0.0f;

    VolumeFloat out(dims, 0.0f);

    GpuVolume gpu_in(in);
    GpuVolume gpu_out(out);
    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{0.25f, 0.0f, 0.0f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.25f));

    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{0.0f, 0.1f, 0.0f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.1f));

    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{0.0f, 0.0f, 0.05f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.05f));
    
    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{10.05f, 0.0f, 0.0f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.0f));
    
    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{0.0f, 10.05f, 0.0f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.0f));

    linear_at_border_kernel<float><<<dim3{1,1,1}, dims>>>(gpu_in, gpu_out, dims, float3{0.0f, 0.0f, 10.05f});

    gpu_out.download(out);
    REQUIRE(out(0,0,0) == Approx(0.0f));
}

